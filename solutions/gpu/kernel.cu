#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <iostream>
#include <ctime>
 
#include "KernelAPI.h"
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void initRand(hiprandState_t *states, unsigned int seed)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &states[tid]);
}

__device__ int rouletteSelectTree(hiprandState_t *states, float *probs, int length)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0;
	
	for (int i = 0; i < length; i++)
	{
		sum += probs[i];
	}
	
	if (sum < 0.001)
	{
	//	printf("hi ");
		return 0;
	}

	float rand = hiprand_uniform(&states[idx]) * sum * 0.999;
	//printf("%f ", rand);
/*	if (threadIdx.x == 0) {
		printf("SUM: %f RAND: %f \n", sum, rand);
	}*/
	float x = 0;
	for (int i = 0; i < length; i++)
	{
		x += probs[i];
/*		if (threadIdx.x == 0)
		{
			printf("PROB: %f PROBSUM: %f \n", probs[i], x);
		}*/
		if (x > rand)
		{
			return i;
		}
	}
	printf("REACHED UNREACHABLE SUM: %f  RAND: %f  X: %f\n", sum, rand, x);
	// shouldn't reach this place
	return 0;
}

__device__ float calcProbability(int *dists, float  *pheroms, int treeIndex, int *cutValues, int diameter)
{
	return powf(pheroms[treeIndex], ALPHA) * powf(1.0f / (dists[treeIndex] + diameter), BETA) * powf(cutValues[treeIndex * 2], GAMMA);
}

__device__ int cutTree(Tree *trees, bool *treeState, int i, int direction, int mapSize, int *forest)
{
	int height = trees[i].h;
	int diameter = trees[i].d;
	int volume = height * diameter;
	int weight = volume * trees[i].c;
	int sum = volume * trees[i].p;
	int x = trees[i].x;
	int y = trees[i].y;
	int cur_tree = i;
	treeState[cur_tree] = false;
	int dx, dy;
	if (direction == 0)
	{
		dx = 0;
		dy = -1;
	}
	if (direction == 1)
	{
		dx = 1;
		dy = 0;
	}
	if (direction == 2)
	{
		dx = 0;
		dy = 1;
	}
	if (direction == 3)
	{
		dx = -1;
		dy = 0;
	}
	height--;
	while (height > 0)
	{
		x += dx;
		y += dy;
		if (x < 0 || y < 0 || x >= mapSize || y >= mapSize)
			break;
		int treeIndex = forest[y * mapSize + x];
		if (treeIndex > 0 && treeState[treeIndex])
		{
			Tree t = trees[treeIndex];
			int w = t.h * t.d * t.c;
			if (weight > w)
			{
				sum += t.h * t.d * t.p;
				height = trees[treeIndex].h;
				weight = w;
				cur_tree = treeIndex;
				treeState[cur_tree] = false;
			}
			else
				break;
		}
		height--;
	}
	return sum;
}

__global__ void runAnt(int treeCount, int timeLimit, int mapSize, Tree *trees, int *distances, float* pheromones, int *cutValues, hiprandState_t *randStates, int *gpuPaths, int *scores, int *lengths, int *forest)
{
	__shared__ bool treeState[MAX_TREES]; // 0 for cut, 1 for available
	__shared__ int currentTree;
	__shared__ int currentTime;
	__shared__ int currentScore;
	__shared__ int numberVisited;
	__shared__ bool theEnd;
	__shared__ float sharedTreeProb[THREADS_PER_ANT]; 
	__shared__ int sharedTreeIdx[THREADS_PER_ANT];
	
	// cache 
//	__shared__ int localDistances[MAX_TREES];
//	__shared__ float localPheromones[MAX_TREES];
	//__shared__ float localCutValues[MAX_TREES*2];
	
	
	int treesPerThread = (treeCount + THREADS_PER_ANT - 1) / THREADS_PER_ANT;
	const int firstTree = threadIdx.x * treesPerThread;
	
	if (firstTree >= treeCount)
	{
		sharedTreeProb[threadIdx.x] = 0;
		return;
	}
	if (firstTree + treesPerThread > treeCount)
	{
		treesPerThread = treeCount - firstTree;
	}
	// Each thread chooses tree from it's own set, then main thread chooses one from these
	float *localTreeProb = new float[treesPerThread];

	// init tree states
	for (int i = firstTree; i < firstTree + treesPerThread; i++)
	{
		treeState[i] = true;
	}
	
	if (threadIdx.x == 0)
	{
		treeState[0] = false;
		currentTree = 0;
		currentTime = 0;
		numberVisited = 0; 
		currentScore = 0;
		theEnd = false;
	}
	__syncthreads();
	//int count = 100; // for safety / debugging
	while (!theEnd)
	{
		// wait for data transfers to complete
/* 	 if (threadIdx.x == 0)
		{
			printf("\n");
		}*/
		//__syncthreads();
		float sum = 0;
		for (int i = 0; i < treesPerThread; i++)
		{
			int tree = i + firstTree;
			int diameter = trees[tree].d;
			if (distances[treeCount*currentTree + tree] + diameter + currentTime <= timeLimit && treeState[tree]) // have enough time to cut and is not yet cut
			{
				localTreeProb[i] = calcProbability(&distances[treeCount*currentTree], &pheromones[treeCount*currentTree], tree, cutValues, diameter);
				sum += localTreeProb[i];
			} else
			{
				localTreeProb[i] = 0.0f;
			}
			//printf("%.2f ", localTreeProb[i]);
			
		}
	/*	if (treesPerThread > 0)
			printf("%.2f ", sum);*/
		
		int localSelectedTree = rouletteSelectTree(randStates, localTreeProb, treesPerThread);
		
		sharedTreeProb[threadIdx.x] = localTreeProb[localSelectedTree];
		sharedTreeIdx[threadIdx.x] = firstTree + localSelectedTree;
		__syncthreads();

		if (threadIdx.x == 0)
		{
			int nextIndex = rouletteSelectTree(randStates, sharedTreeProb, THREADS_PER_ANT);
			if (sharedTreeIdx[nextIndex] == 0)
			{
				// tour finished (end of game time)
	//			printf("Finishing ");
				theEnd = true;
				//break;
			}
			else {
				currentTime += distances[treeCount*currentTree + sharedTreeIdx[nextIndex]] + trees[sharedTreeIdx[nextIndex]].d;
				currentTree = sharedTreeIdx[nextIndex];

				gpuPaths[blockIdx.x * treeCount + numberVisited * 2] = currentTree;
				
				int dir = cutValues[currentTree * 2 + 1];

				gpuPaths[blockIdx.x * treeCount + numberVisited * 2 + 1] = dir;
				currentScore += cutTree(trees, treeState, currentTree, dir, mapSize, forest);
				numberVisited++;
			}
		}

		__syncthreads();
	}
	if (threadIdx.x == 0) {
		scores[blockIdx.x] = currentScore;
		lengths[blockIdx.x] = numberVisited;
	}
	delete[] localTreeProb;
}

hiprandState_t *randStates;
Tree *gpuTrees;
int *gpuDistances;
float *gpuPheromones;
int *gpuCutValues;
int *gpuPaths;
int *gpuScores;
int *gpuLengths;
int *gpuForest;
const int threadsPerBlock = THREADS_PER_ANT;
const int blocks = ANTS;


void initGPU(int numberOfTrees, int mapSize, Tree *trees, int *distances, int *cutValues, int *forest)
{
	hipMalloc((void**)&randStates, sizeof(hiprandState_t) * threadsPerBlock * blocks);

	initRand << <blocks, threadsPerBlock >> >(randStates, time(0));

	hipMalloc((void**)&gpuTrees, sizeof(Tree) * numberOfTrees);
	hipMemcpy(gpuTrees, trees, sizeof(Tree)*numberOfTrees, hipMemcpyHostToDevice);

	hipMalloc((void**)&gpuDistances, sizeof(int) * numberOfTrees*numberOfTrees);
	hipMemcpy(gpuDistances, distances, sizeof(int)*numberOfTrees*numberOfTrees, hipMemcpyHostToDevice);

	hipMalloc((void**)&gpuPheromones, sizeof(float)*numberOfTrees*numberOfTrees);

	hipMalloc((void**)&gpuCutValues, sizeof(int)*numberOfTrees*2);
	hipMemcpy(gpuCutValues, cutValues, sizeof(int)*numberOfTrees * 2, hipMemcpyHostToDevice);

	hipMalloc((void**)&gpuPaths, sizeof(int) * ANTS * numberOfTrees); // STORES INDICES AND DIRECTIONS; SIZE IS ENOUGH FOR HALF TREES-1

	hipMalloc((void**)&gpuScores, sizeof(int) * ANTS);
	hipMalloc((void**)&gpuLengths, sizeof(int) * ANTS);

	hipMalloc((void**)&gpuForest, sizeof(int)*mapSize*mapSize);
	hipMemcpy(gpuForest, forest, sizeof(int)*mapSize*mapSize, hipMemcpyHostToDevice);

}

void cleanupGPU()
{
	hipFree(randStates);
	hipFree(gpuTrees);
	hipFree(gpuDistances);
	hipFree(gpuPheromones);
	hipFree(gpuCutValues);
	hipFree(gpuPaths);
	hipFree(gpuScores);
	hipFree(gpuForest);
	hipFree(gpuLengths);
}

int runKernel(int numberOfTrees, float *pheromones, int timeLimit, int mapSize, int *resultScores, int *resultPaths, int *resultLengths)
{	
	hipMemcpy(gpuPheromones, pheromones, sizeof(float)*numberOfTrees*numberOfTrees, hipMemcpyHostToDevice);

	
	runAnt << < blocks, threadsPerBlock >> >(numberOfTrees, timeLimit, mapSize, gpuTrees, gpuDistances, gpuPheromones, gpuCutValues, randStates, gpuPaths, gpuScores, gpuLengths, gpuForest);


	hipMemcpy(resultScores, gpuScores, sizeof(int)*ANTS, hipMemcpyDeviceToHost);
	hipMemcpy(resultLengths, gpuLengths, sizeof(int)*ANTS, hipMemcpyDeviceToHost);
	hipMemcpy(resultPaths, gpuPaths, sizeof(int)*ANTS*numberOfTrees, hipMemcpyDeviceToHost);

	return 0;
}

/*   const int arraySize = 5;
const int a[arraySize] = { 1, 2, 3, 4, 5 };
const int b[arraySize] = { 10, 20, 30, 40, 50 };
int c[arraySize] = { 0 };

// Add vectors in parallel.
hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addWithCuda failed!");
return 1;
}

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceReset failed!");
return 1;
} */

// Helper function for using CUDA to add vectors in parallel. 
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
} */
